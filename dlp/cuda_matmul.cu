\
 // cuda_matmul.cu
 // Compile with: nvcc -O3 cuda_matmul.cu -o cuda_matmul
 #include <hip/hip_runtime.h>
 #include <stdio.h>
 #include <stdlib.h>

 __global__ void matmul_kernel(float* A, float* B, float* C, int N) {
     int row = blockIdx.y * blockDim.y + threadIdx.y;
     int col = blockIdx.x * blockDim.x + threadIdx.x;
     if (row < N && col < N) {
         float sum = 0.0f;
         for (int k=0;k<N;k++) sum += A[row*N + k] * B[k*N + col];
         C[row*N + col] = sum;
     }
 }

 int main(int argc, char** argv) {
     int N = 256;
     if (argc > 1) N = atoi(argv[1]);
     size_t bytes = N * N * sizeof(float);
     float *h_A = (float*) malloc(bytes);
     float *h_B = (float*) malloc(bytes);
     float *h_C = (float*) malloc(bytes);
     for (int i=0;i<N*N;i++) { h_A[i]=1.0f; h_B[i]=1.0f; }

     float *d_A, *d_B, *d_C;
     hipMalloc(&d_A, bytes); hipMalloc(&d_B, bytes); hipMalloc(&d_C, bytes);
     hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
     hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

     dim3 block(16,16); dim3 grid((N+15)/16, (N+15)/16);
     matmul_kernel<<<grid, block>>>(d_A, d_B, d_C, N);
     hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

     printf("C[0]=%f C[N*N-1]=%f\n", h_C[0], h_C[N*N-1]);
     hipFree(d_A); hipFree(d_B); hipFree(d_C);
     free(h_A); free(h_B); free(h_C);
     return 0;
 }
